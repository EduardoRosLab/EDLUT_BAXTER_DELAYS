#include "hip/hip_runtime.h"
/***************************************************************************
 *                           LIFTimeDrivenModel_1_2_GPU.cu                 *
 *                           -------------------                           *
 * copyright            : (C) 2012 by Francisco Naveros                    *
 * email                : fnaveros@ugr.es                                  *
 ***************************************************************************/

/***************************************************************************
 *                                                                         *
 *   This program is free software; you can redistribute it and/or modify  *
 *   it under the terms of the GNU General Public License as published by  *
 *   the Free Software Foundation; either version 3 of the License, or     *
 *   (at your option) any later version.                                   *
 *                                                                         *
 ***************************************************************************/

#include "../../include/neuron_model/HHTimeDrivenModel_GPU.h"
#include "../../include/neuron_model/HHTimeDrivenModel_GPU2.h"
#include "../../include/neuron_model/VectorNeuronState.h"
#include "../../include/neuron_model/VectorNeuronState_GPU.h"

#include <iostream>
#include <cmath>
#include <string>

#include "../../include/spike/EDLUTFileException.h"
#include "../../include/spike/Neuron.h"
#include "../../include/spike/InternalSpike.h"
#include "../../include/spike/PropagatedSpike.h"
#include "../../include/spike/Interconnection.h"

#include "../../include/simulation/Utils.h"

#include "../../include/openmp/openmp.h"

#include "../../include/hipError_t.h"
//Library for CUDA
#include "hip/hip_runtime.h"
#include ""

void HHTimeDrivenModel_GPU::LoadNeuronModel(string ConfigFile) throw (EDLUTFileException){
	FILE *fh;
	long Currentline = 0L;
	fh=fopen(ConfigFile.c_str(),"rt");
	if(fh){
		Currentline = 1L;
		skip_comments(fh, Currentline);
		if (fscanf(fh, "%f", &this->eexc) == 1){
			skip_comments(fh, Currentline);
			if (fscanf(fh, "%f", &this->einh) == 1){
				skip_comments(fh, Currentline);
				if (fscanf(fh, "%f", &this->erest) == 1){
					skip_comments(fh, Currentline);
					if (fscanf(fh, "%f", &this->grest) == 1 && this->grest > 0.0f){
						skip_comments(fh, Currentline);
						if (fscanf(fh, "%f", &this->cm) == 1 && this->cm > 0.0f){
							skip_comments(fh, Currentline);
							if (fscanf(fh, "%f", &this->vthr) == 1){
								skip_comments(fh, Currentline);
								if (fscanf(fh, "%f", &this->texc) == 1 && this->texc > 0.0f){
									skip_comments(fh, Currentline);
									if (fscanf(fh, "%f", &this->tinh) == 1 && this->tinh > 0.0f){
										skip_comments(fh, Currentline);
										if (fscanf(fh, "%f", &this->gNa) == 1 && this->gNa > 0.0f){
											skip_comments(fh, Currentline);
											if (fscanf(fh, "%f", &this->gKd) == 1 && this->gKd > 0.0f){
												skip_comments(fh, Currentline);
												if (fscanf(fh, "%f", &this->ENa) == 1){
													skip_comments(fh, Currentline);
													if (fscanf(fh, "%f", &this->EK) == 1){
														skip_comments(fh, Currentline);
														if (fscanf(fh, "%f", &this->VT) == 1){

															this->State = (VectorNeuronState_GPU *) new VectorNeuronState_GPU(N_NeuronStateVariables);

														}else {
															throw EDLUTFileException(TASK_HH_TIME_DRIVEN_MODEL_GPU_LOAD, ERROR_HH_TIME_DRIVEN_MODEL_VT, REPAIR_NEURON_MODEL_VALUES, Currentline, ConfigFile.c_str(), true);
														}
													}else {
														throw EDLUTFileException(TASK_HH_TIME_DRIVEN_MODEL_GPU_LOAD, ERROR_HH_TIME_DRIVEN_MODEL_EK, REPAIR_NEURON_MODEL_VALUES, Currentline, ConfigFile.c_str(), true);
													}
												}else {
													throw EDLUTFileException(TASK_HH_TIME_DRIVEN_MODEL_GPU_LOAD, ERROR_HH_TIME_DRIVEN_MODEL_ENA, REPAIR_NEURON_MODEL_VALUES, Currentline, ConfigFile.c_str(), true);
												}
											}else {
												throw EDLUTFileException(TASK_HH_TIME_DRIVEN_MODEL_GPU_LOAD, ERROR_HH_TIME_DRIVEN_MODEL_GKD, REPAIR_NEURON_MODEL_VALUES, Currentline, ConfigFile.c_str(), true);
											}
										}else {
											throw EDLUTFileException(TASK_HH_TIME_DRIVEN_MODEL_GPU_LOAD, ERROR_HH_TIME_DRIVEN_MODEL_GNA, REPAIR_NEURON_MODEL_VALUES, Currentline, ConfigFile.c_str(), true);
										}
									}else {
										throw EDLUTFileException(TASK_HH_TIME_DRIVEN_MODEL_GPU_LOAD, ERROR_HH_TIME_DRIVEN_MODEL_TINH, REPAIR_NEURON_MODEL_VALUES, Currentline, ConfigFile.c_str(), true);
									}
								}else {
									throw EDLUTFileException(TASK_HH_TIME_DRIVEN_MODEL_GPU_LOAD, ERROR_HH_TIME_DRIVEN_MODEL_TEXC, REPAIR_NEURON_MODEL_VALUES, Currentline, ConfigFile.c_str(), true);
								}
							}else {
								throw EDLUTFileException(TASK_HH_TIME_DRIVEN_MODEL_GPU_LOAD, ERROR_HH_TIME_DRIVEN_MODEL_VTHR, REPAIR_NEURON_MODEL_VALUES, Currentline, ConfigFile.c_str(), true);
							}
						}else {
							throw EDLUTFileException(TASK_HH_TIME_DRIVEN_MODEL_GPU_LOAD, ERROR_HH_TIME_DRIVEN_MODEL_CM, REPAIR_NEURON_MODEL_VALUES, Currentline, ConfigFile.c_str(), true);
						}
					}else {
						throw EDLUTFileException(TASK_HH_TIME_DRIVEN_MODEL_GPU_LOAD, ERROR_HH_TIME_DRIVEN_MODEL_GREST, REPAIR_NEURON_MODEL_VALUES, Currentline, ConfigFile.c_str(), true);
					}
				}else {
					throw EDLUTFileException(TASK_HH_TIME_DRIVEN_MODEL_GPU_LOAD, ERROR_HH_TIME_DRIVEN_MODEL_EREST, REPAIR_NEURON_MODEL_VALUES, Currentline, ConfigFile.c_str(), true);
				}
			}else {
				throw EDLUTFileException(TASK_HH_TIME_DRIVEN_MODEL_GPU_LOAD, ERROR_HH_TIME_DRIVEN_MODEL_EINH, REPAIR_NEURON_MODEL_VALUES, Currentline, ConfigFile.c_str(), true);
			}
		}else {
			throw EDLUTFileException(TASK_HH_TIME_DRIVEN_MODEL_GPU_LOAD, ERROR_HH_TIME_DRIVEN_MODEL_EEXC, REPAIR_NEURON_MODEL_VALUES, Currentline, ConfigFile.c_str(), true);
		}

  		//INTEGRATION METHOD
		this->integrationMethod_GPU = LoadIntegrationMethod_GPU::loadIntegrationMethod_GPU((TimeDrivenNeuronModel_GPU *)this, this->GetModelID(), fh, &Currentline, N_NeuronStateVariables, N_DifferentialNeuronState, N_TimeDependentNeuronState);

		//SET TIME-DRIVEN STEP SIZE
		this->SetTimeDrivenStepSize(this->integrationMethod_GPU->elapsedTimeInSeconds);
	}
	else{
		throw EDLUTFileException(TASK_HH_TIME_DRIVEN_MODEL_GPU_LOAD, ERROR_NEURON_MODEL_OPEN, REPAIR_NEURON_MODEL_NAME, Currentline, ConfigFile.c_str(), true);
	}
	fclose(fh);
}

HHTimeDrivenModel_GPU::HHTimeDrivenModel_GPU(string NeuronTypeID, string NeuronModelID): TimeDrivenNeuronModel_GPU(NeuronTypeID, NeuronModelID, MilisecondScale){
	//eexc=0.0f; //mV
	//einh=-80.0f; //mV
	//erest=-65.0f; //mV
	//vthr=-30.0f; //mV
	//cm=120.0f; //pF   
	//texc=5.0f;//ms
	//tinh=10.0f;//ms
	//grest=10.0f;//nS
	//gNa=20000.0;//nS
	//gKd=6000.0f;//nS
	//ENa=50.0f;//mV
	//EK=-90.0f;//mV
	//VT=-52.0f;//mV

}

HHTimeDrivenModel_GPU::~HHTimeDrivenModel_GPU(void){
	DeleteClassGPU2();
}

void HHTimeDrivenModel_GPU::LoadNeuronModel() throw (EDLUTFileException){
	this->LoadNeuronModel(this->GetModelID()+".cfg");
}

VectorNeuronState * HHTimeDrivenModel_GPU::InitializeState(){
	return this->GetVectorNeuronState();
}


InternalSpike * HHTimeDrivenModel_GPU::ProcessInputSpike(Interconnection * inter, double time){
	this->State_GPU->AuxStateCPU[inter->GetType()*State_GPU->GetSizeState() + inter->GetTargetNeuronModelIndex()] += inter->GetWeight();

	return 0;
}


__global__ void HHTimeDrivenModel_GPU_UpdateState(HHTimeDrivenModel_GPU2 ** NeuronModel_GPU2, double CurrentTime){
	(*NeuronModel_GPU2)->UpdateState(CurrentTime);
}

		
bool HHTimeDrivenModel_GPU::UpdateState(int index, double CurrentTime){
	VectorNeuronState_GPU *state = (VectorNeuronState_GPU *) State;

	//----------------------------------------------
	if(prop.canMapHostMemory){
		HHTimeDrivenModel_GPU_UpdateState<<<N_block,N_thread>>>(NeuronModel_GPU2, CurrentTime);
	}else{
		HANDLE_ERROR(hipMemcpy(state->AuxStateGPU,state->AuxStateCPU,this->N_TimeDependentNeuronState*state->SizeStates*sizeof(float),hipMemcpyHostToDevice));
		HHTimeDrivenModel_GPU_UpdateState<<<N_block,N_thread>>>(NeuronModel_GPU2, CurrentTime);
		HANDLE_ERROR(hipMemcpy(state->InternalSpikeCPU,state->InternalSpikeGPU,state->SizeStates*sizeof(bool),hipMemcpyDeviceToHost));
	}


	if(this->GetVectorNeuronState()->Get_Is_Monitored()){
		HANDLE_ERROR(hipMemcpy(state->VectorNeuronStates,state->VectorNeuronStates_GPU,state->GetNumberOfVariables()*state->SizeStates*sizeof(float),hipMemcpyDeviceToHost));
		HANDLE_ERROR(hipMemcpy(state->LastUpdate,state->LastUpdateGPU,state->SizeStates*sizeof(double),hipMemcpyDeviceToHost));
		HANDLE_ERROR(hipMemcpy(state->LastSpikeTime,state->LastSpikeTimeGPU,state->SizeStates*sizeof(double),hipMemcpyDeviceToHost));
	}
 

	HANDLE_ERROR(hipEventRecord(stop, 0)); 
	HANDLE_ERROR(hipEventSynchronize(stop));


	memset(state->AuxStateCPU,0,N_TimeDependentNeuronState*state->SizeStates*sizeof(float));

	return false;

}


enum NeuronModelOutputActivityType HHTimeDrivenModel_GPU::GetModelOutputActivityType(){
	return OUTPUT_SPIKE;
}

enum NeuronModelInputActivityType HHTimeDrivenModel_GPU::GetModelInputActivityType(){
	return INPUT_SPIKE;
}


ostream & HHTimeDrivenModel_GPU::PrintInfo(ostream & out){
	return out;
}	


void HHTimeDrivenModel_GPU::InitializeStates(int N_neurons, int OpenMPQueueIndex){

	//Select the correnpondent device. 
	HANDLE_ERROR(hipSetDevice(GPUsIndex[OpenMPQueueIndex % NumberOfGPUs]));  
	HANDLE_ERROR(hipEventCreate(&stop));
	HANDLE_ERROR(hipGetDeviceProperties( &prop, GPUsIndex[OpenMPQueueIndex % NumberOfGPUs]));


	this->State_GPU = (VectorNeuronState_GPU *) this->State;
	
	//Initialize neural state variables.
	//m
	float alpha_m=0.32f*(13.0f-erest+VT)/(exp((13.0f-erest+VT)/4.0f)-1.0f);
	float beta_m=0.28f*(erest-VT-40.0f)/(exp((erest-VT-40.0f)/5.0f)-1.0f);
	float m_inf=alpha_m/(alpha_m+beta_m);

	//h
	float alpha_h=0.128f*exp((17.0f-erest+VT)/18.0f);
	float beta_h=4.0f/(1.0f+exp((40.0f-erest+VT)/5.0f));
	float h_inf=alpha_h/(alpha_h+beta_h);


	//n
	float alpha_n=0.032f*(15.0f-erest+VT)/(exp((15.0f-erest+VT)/5.0f)-1.0f);
	float beta_n=0.5f*exp((10.0f-erest+VT)/40.0f);
	float n_inf=alpha_n/(alpha_n+beta_n);

	float initialization[] = {erest,m_inf, h_inf, n_inf, 0.0f, 0.0f};

	State_GPU->InitializeStatesGPU(N_neurons, initialization, N_TimeDependentNeuronState, prop);

	//INITIALIZE CLASS IN GPU
	this->InitializeClassGPU2(N_neurons);


	InitializeVectorNeuronState_GPU2();
}




__global__ void HHTimeDrivenModel_GPU_InitializeClassGPU2(HHTimeDrivenModel_GPU2 ** NeuronModel_GPU2, 
		float new_eexc, float new_einh, float new_erest, float new_grest, float new_cm, float new_vthr, float new_texc, 
		float new_tinh, float new_gNa, float new_gKd, float new_ENa, float new_EK, float new_VT,
		char const* integrationName, int N_neurons, void ** Buffer_GPU){
	if(blockIdx.x==0 && threadIdx.x==0){
		(*NeuronModel_GPU2)=new HHTimeDrivenModel_GPU2(new_eexc, new_einh, new_erest, new_grest, new_cm, new_vthr, new_texc, 
				new_tinh, new_gNa, new_gKd, new_ENa, new_EK, new_VT, integrationName, N_neurons, Buffer_GPU);
	}
}

void HHTimeDrivenModel_GPU::InitializeClassGPU2(int N_neurons){
	hipMalloc(&NeuronModel_GPU2, sizeof(HHTimeDrivenModel_GPU **));
	
	char * integrationNameGPU;
	hipMalloc((void **)&integrationNameGPU,32*4);
	HANDLE_ERROR(hipMemcpy(integrationNameGPU,integrationMethod_GPU->GetType(),32*4,hipMemcpyHostToDevice));

	this->N_thread = 128;
	this->N_block=prop.multiProcessorCount*16;
	if((N_neurons+N_thread-1)/N_thread < N_block){
		N_block = (N_neurons+N_thread-1)/N_thread;
	}
	int Total_N_thread=N_thread*N_block;

	integrationMethod_GPU->InitializeMemoryGPU(N_neurons, Total_N_thread);

	HHTimeDrivenModel_GPU_InitializeClassGPU2<<<1,1>>>(NeuronModel_GPU2, eexc, einh, erest, grest, cm, vthr, texc, 
		tinh, gNa, gKd, ENa, EK, VT, integrationNameGPU, N_neurons, integrationMethod_GPU->Buffer_GPU);

	hipFree(integrationNameGPU);
}



__global__ void initializeVectorNeuronState_GPU2(HHTimeDrivenModel_GPU2 ** NeuronModel_GPU2, int NumberOfVariables, float * InitialStateGPU, float * AuxStateGPU, float * StateGPU, double * LastUpdateGPU, double * LastSpikeTimeGPU, bool * InternalSpikeGPU, int SizeStates){
	if(blockIdx.x==0 && threadIdx.x==0){
		(*NeuronModel_GPU2)->InitializeVectorNeuronState_GPU2(NumberOfVariables, InitialStateGPU, AuxStateGPU, StateGPU, LastUpdateGPU, LastSpikeTimeGPU, InternalSpikeGPU, SizeStates);
	}
}

void HHTimeDrivenModel_GPU::InitializeVectorNeuronState_GPU2(){
	VectorNeuronState_GPU *state = (VectorNeuronState_GPU *) State;
	initializeVectorNeuronState_GPU2<<<1,1>>>(NeuronModel_GPU2, state->NumberOfVariables, state->InitialStateGPU, state->AuxStateGPU, state->VectorNeuronStates_GPU, state->LastUpdateGPU, state->LastSpikeTimeGPU, state->InternalSpikeGPU, state->SizeStates);
}


__global__ void DeleteClass_GPU2(HHTimeDrivenModel_GPU2 ** NeuronModel_GPU2){
	if(blockIdx.x==0 && threadIdx.x==0){
		delete (*NeuronModel_GPU2); 
	}
}


void HHTimeDrivenModel_GPU::DeleteClassGPU2(){
    DeleteClass_GPU2<<<1,1>>>(NeuronModel_GPU2);
    hipFree(NeuronModel_GPU2);
}


bool HHTimeDrivenModel_GPU::CheckSynapseType(Interconnection * connection){
	int Type = connection->GetType();
	if (Type<N_TimeDependentNeuronState && Type >= 0){
		NeuronModel * model = connection->GetSource()->GetNeuronModel();
		//Synapse types that process input spikes 
		if (Type < N_TimeDependentNeuronState && model->GetModelOutputActivityType() == OUTPUT_SPIKE)
			return true;
		else{
			cout << "Synapses type " << Type << " of neuron model " << this->GetTypeID() << ", " << this->GetModelID() << " must receive spikes. The source model generates currents." << endl;
			return false;
		}
		//Synapse types that process input current 
	}
	else{
		cout << "Neuron model " << this->GetTypeID() << ", " << this->GetModelID() << " does not support input synapses of type " << Type << ". Just defined " << N_TimeDependentNeuronState << " synapses types." << endl;
		return false;
	}
}