#include "hip/hip_runtime.h"
/***************************************************************************
 *                           LIFTimeDrivenModel_1_4_GPU_NEW.cu             *
 *                           -------------------                           *
 * copyright            : (C) 2016 by Francisco Naveros                    *
 * email                : fnaveros@ugr.es                                  *
 ***************************************************************************/

/***************************************************************************
 *                                                                         *
 *   This program is free software; you can redistribute it and/or modify  *
 *   it under the terms of the GNU General Public License as published by  *
 *   the Free Software Foundation; either version 3 of the License, or     *
 *   (at your option) any later version.                                   *
 *                                                                         *
 ***************************************************************************/

#include "../../include/neuron_model/LIFTimeDrivenModel_1_4_GPU_NEW.h"
#include "../../include/neuron_model/LIFTimeDrivenModel_1_4_GPU2_NEW.h"
#include "../../include/neuron_model/VectorNeuronState.h"
#include "../../include/neuron_model/VectorNeuronState_GPU.h"

#include <iostream>
#include <cmath>
#include <string>

#include "../../include/spike/EDLUTFileException.h"
#include "../../include/spike/Neuron.h"
#include "../../include/spike/InternalSpike.h"
#include "../../include/spike/PropagatedSpike.h"
#include "../../include/spike/Interconnection.h"

#include "../../include/simulation/Utils.h"

#include "../../include/openmp/openmp.h"

#include "../../include/hipError_t.h"
//Library for CUDA
//#include <hip/hip_runtime_api.h>
#include "hip/hip_runtime.h"
#include ""

void LIFTimeDrivenModel_1_4_GPU_NEW::LoadNeuronModel(string ConfigFile) throw (EDLUTFileException){
	FILE *fh;
	long Currentline = 0L;
	fh=fopen(ConfigFile.c_str(),"rt");
	if(fh){
		Currentline=1L;
		skip_comments(fh,Currentline);
		if(fscanf(fh,"%f",&this->eexc)==1){
			skip_comments(fh,Currentline);

			if (fscanf(fh,"%f",&this->einh)==1){
				skip_comments(fh,Currentline);

				if(fscanf(fh,"%f",&this->erest)==1){
					skip_comments(fh,Currentline);

					if(fscanf(fh,"%f",&this->vthr)==1){
						skip_comments(fh,Currentline);

						if(fscanf(fh,"%f",&this->cm)==1 && this->cm > 0.0f){
							skip_comments(fh,Currentline);

							if(fscanf(fh,"%f",&this->tampa)==1 && this->tampa > 0.0f){
								skip_comments(fh,Currentline);

								if(fscanf(fh,"%f",&this->tnmda)==1 && this->tnmda > 0.0f){
									skip_comments(fh,Currentline);
									
									if(fscanf(fh,"%f",&this->tinh)==1 && this->tinh > 0.0f){
										skip_comments(fh,Currentline);

										if(fscanf(fh,"%f",&this->tgj)==1 && this->tgj > 0.0f){
											skip_comments(fh,Currentline);
											if(fscanf(fh,"%f",&this->tref)==1 && this->tref > 0.0f){
												skip_comments(fh,Currentline);

												if(fscanf(fh,"%f",&this->grest)==1 && this->grest > 0.0f){
													skip_comments(fh,Currentline);

													if(fscanf(fh,"%f",&this->fgj)==1){
														skip_comments(fh,Currentline);


														this->State = (VectorNeuronState_GPU *) new VectorNeuronState_GPU(N_NeuronStateVariables);

													}else {
														throw EDLUTFileException(TASK_LIF_TIME_DRIVEN_MODEL_1_4_GPU_NEW_LOAD, ERROR_LIF_TIME_DRIVEN_MODEL_1_4_NEW_FGJ, REPAIR_NEURON_MODEL_VALUES, Currentline, ConfigFile.c_str(), true);
													}
												}else {
													throw EDLUTFileException(TASK_LIF_TIME_DRIVEN_MODEL_1_4_GPU_NEW_LOAD, ERROR_LIF_TIME_DRIVEN_MODEL_1_4_NEW_GREST, REPAIR_NEURON_MODEL_VALUES, Currentline, ConfigFile.c_str(), true);
												}
											}else {
												throw EDLUTFileException(TASK_LIF_TIME_DRIVEN_MODEL_1_4_GPU_NEW_LOAD, ERROR_LIF_TIME_DRIVEN_MODEL_1_4_NEW_TREF, REPAIR_NEURON_MODEL_VALUES, Currentline, ConfigFile.c_str(), true);
											}
										}else {
											throw EDLUTFileException(TASK_LIF_TIME_DRIVEN_MODEL_1_4_GPU_NEW_LOAD, ERROR_LIF_TIME_DRIVEN_MODEL_1_4_NEW_GAP, REPAIR_NEURON_MODEL_VALUES, Currentline, ConfigFile.c_str(), true);
										}
									}else {
										throw EDLUTFileException(TASK_LIF_TIME_DRIVEN_MODEL_1_4_GPU_NEW_LOAD, ERROR_LIF_TIME_DRIVEN_MODEL_1_4_NEW_TINH, REPAIR_NEURON_MODEL_VALUES, Currentline, ConfigFile.c_str(), true);
									}
								}else {
									throw EDLUTFileException(TASK_LIF_TIME_DRIVEN_MODEL_1_4_GPU_NEW_LOAD, ERROR_LIF_TIME_DRIVEN_MODEL_1_4_NEW_TNMDA, REPAIR_NEURON_MODEL_VALUES, Currentline, ConfigFile.c_str(), true);
								}
							}else {
								throw EDLUTFileException(TASK_LIF_TIME_DRIVEN_MODEL_1_4_GPU_NEW_LOAD, ERROR_LIF_TIME_DRIVEN_MODEL_1_4_NEW_TAMPA, REPAIR_NEURON_MODEL_VALUES, Currentline, ConfigFile.c_str(), true);
							}
						}else {
							throw EDLUTFileException(TASK_LIF_TIME_DRIVEN_MODEL_1_4_GPU_NEW_LOAD, ERROR_LIF_TIME_DRIVEN_MODEL_1_4_NEW_CM, REPAIR_NEURON_MODEL_VALUES, Currentline, ConfigFile.c_str(), true);
						}
					}else {
						throw EDLUTFileException(TASK_LIF_TIME_DRIVEN_MODEL_1_4_GPU_NEW_LOAD, ERROR_LIF_TIME_DRIVEN_MODEL_1_4_NEW_VTHR, REPAIR_NEURON_MODEL_VALUES, Currentline, ConfigFile.c_str(), true);
					}
				}else {
					throw EDLUTFileException(TASK_LIF_TIME_DRIVEN_MODEL_1_4_GPU_NEW_LOAD, ERROR_LIF_TIME_DRIVEN_MODEL_1_4_NEW_EREST, REPAIR_NEURON_MODEL_VALUES, Currentline, ConfigFile.c_str(), true);
				}
			}else {
				throw EDLUTFileException(TASK_LIF_TIME_DRIVEN_MODEL_1_4_GPU_NEW_LOAD, ERROR_LIF_TIME_DRIVEN_MODEL_1_4_NEW_EINH, REPAIR_NEURON_MODEL_VALUES, Currentline, ConfigFile.c_str(), true);
			}
		}else {
			throw EDLUTFileException(TASK_LIF_TIME_DRIVEN_MODEL_1_4_GPU_NEW_LOAD, ERROR_LIF_TIME_DRIVEN_MODEL_1_4_NEW_EEXC, REPAIR_NEURON_MODEL_VALUES, Currentline, ConfigFile.c_str(), true);
		}

  		//INTEGRATION METHOD
		this->integrationMethod_GPU = LoadIntegrationMethod_GPU::loadIntegrationMethod_GPU((TimeDrivenNeuronModel_GPU *)this, this->GetModelID(), fh, &Currentline, N_NeuronStateVariables, N_DifferentialNeuronState, N_TimeDependentNeuronState);

		//SET TIME-DRIVEN STEP SIZE
		this->SetTimeDrivenStepSize(this->integrationMethod_GPU->elapsedTimeInSeconds);
	}else{
		throw EDLUTFileException(TASK_LIF_TIME_DRIVEN_MODEL_1_4_GPU_NEW_LOAD, ERROR_NEURON_MODEL_OPEN, REPAIR_NEURON_MODEL_NAME, Currentline, ConfigFile.c_str(), true);
	}
	fclose(fh);
}

LIFTimeDrivenModel_1_4_GPU_NEW::LIFTimeDrivenModel_1_4_GPU_NEW(string NeuronTypeID, string NeuronModelID): TimeDrivenNeuronModel_GPU(NeuronTypeID, NeuronModelID, MilisecondScale), eexc(0), einh(0), erest(0), vthr(0), cm(0), tampa(0), tnmda(0), tinh(0), tgj(0),
		tref(0), grest(0){
}

LIFTimeDrivenModel_1_4_GPU_NEW::~LIFTimeDrivenModel_1_4_GPU_NEW(void){
	DeleteClassGPU2();
}

void LIFTimeDrivenModel_1_4_GPU_NEW::LoadNeuronModel() throw (EDLUTFileException){
	this->LoadNeuronModel(this->GetModelID()+".cfg");
}

VectorNeuronState * LIFTimeDrivenModel_1_4_GPU_NEW::InitializeState(){
	return this->GetVectorNeuronState();
}


InternalSpike * LIFTimeDrivenModel_1_4_GPU_NEW::ProcessInputSpike(Interconnection * inter, double time){
	this->State_GPU->AuxStateCPU[inter->GetType()*State_GPU->GetSizeState() + inter->GetTargetNeuronModelIndex()] += inter->GetWeight();

	return 0;
}


__global__ void LIFTimeDrivenModel_1_4_GPU_NEW_UpdateState(LIFTimeDrivenModel_1_4_GPU2_NEW ** NeuronModel_GPU2, double CurrentTime){
	(*NeuronModel_GPU2)->UpdateState(CurrentTime);
}
		
bool LIFTimeDrivenModel_1_4_GPU_NEW::UpdateState(int index, double CurrentTime){
	
	VectorNeuronState_GPU *state = (VectorNeuronState_GPU *) State;

	//----------------------------------------------
	if(prop.canMapHostMemory){
		LIFTimeDrivenModel_1_4_GPU_NEW_UpdateState<<<N_block,N_thread>>>(NeuronModel_GPU2, CurrentTime);
	}else{
		HANDLE_ERROR(hipMemcpy(state->AuxStateGPU,state->AuxStateCPU,this->N_TimeDependentNeuronState*state->SizeStates*sizeof(float),hipMemcpyHostToDevice));
		LIFTimeDrivenModel_1_4_GPU_NEW_UpdateState<<<N_block,N_thread>>>(NeuronModel_GPU2, CurrentTime);
		HANDLE_ERROR(hipMemcpy(state->InternalSpikeCPU,state->InternalSpikeGPU,state->SizeStates*sizeof(bool),hipMemcpyDeviceToHost));
	}


	if(this->GetVectorNeuronState()->Get_Is_Monitored()){
		HANDLE_ERROR(hipMemcpy(state->VectorNeuronStates,state->VectorNeuronStates_GPU,state->GetNumberOfVariables()*state->SizeStates*sizeof(float),hipMemcpyDeviceToHost));
		HANDLE_ERROR(hipMemcpy(state->LastUpdate,state->LastUpdateGPU,state->SizeStates*sizeof(double),hipMemcpyDeviceToHost));
		HANDLE_ERROR(hipMemcpy(state->LastSpikeTime,state->LastSpikeTimeGPU,state->SizeStates*sizeof(double),hipMemcpyDeviceToHost));
	}
 

	HANDLE_ERROR(hipEventRecord(stop, 0)); 
	HANDLE_ERROR(hipEventSynchronize(stop));


	memset(state->AuxStateCPU,0,N_TimeDependentNeuronState*state->SizeStates*sizeof(float));

	return false;

}


enum NeuronModelOutputActivityType LIFTimeDrivenModel_1_4_GPU_NEW::GetModelOutputActivityType(){
	return OUTPUT_SPIKE;
}

enum NeuronModelInputActivityType LIFTimeDrivenModel_1_4_GPU_NEW::GetModelInputActivityType(){
	return INPUT_SPIKE;
}



ostream & LIFTimeDrivenModel_1_4_GPU_NEW::PrintInfo(ostream & out){
	out << "- Leaky Time-Driven Model: " << this->GetModelID() << endl;

	out << "\tExc. Reversal Potential: " << this->eexc << "mV\tInh. Reversal Potential: " << this->einh << "mV\tResting potential: " << this->erest << "mV" << endl;

	out << "\tFiring threshold: " << this->vthr << "mV\tMembrane capacitance: " << this->cm << "pF\tAMPA Time Constant: " << this->tampa << "ms\tNMDA Time Constant: " << this->tnmda << "ms"<< endl;

	out << "\tInhibitory time constant: " << this->tinh << "ms\tGap junction time constant: " << this->tgj << "ms\tRefractory Period: " << this->tref << "ms\tResting Conductance: " << this->grest << "nS" << endl;

	out << "\tGap junction factor: " << this->fgj << "mV/nS" << endl;

	return out;
}	


void LIFTimeDrivenModel_1_4_GPU_NEW::InitializeStates(int N_neurons, int OpenMPQueueIndex){

	//Select the correnpondent device. 
	HANDLE_ERROR(hipSetDevice(GPUsIndex[OpenMPQueueIndex % NumberOfGPUs]));  
	HANDLE_ERROR(hipEventCreate(&stop));
	HANDLE_ERROR(hipGetDeviceProperties( &prop, GPUsIndex[OpenMPQueueIndex % NumberOfGPUs]));

	VectorNeuronState_GPU * state = (VectorNeuronState_GPU *) this->State;
	
	float initialization[] = {erest,0.0,0.0,0.0,0.0};
	state->InitializeStatesGPU(N_neurons, initialization, N_TimeDependentNeuronState, prop);

	//INITIALIZE CLASS IN GPU
	this->InitializeClassGPU2(N_neurons);


	InitializeVectorNeuronState_GPU2();
}




__global__ void LIFTimeDrivenModel_1_4_GPU_NEW_InitializeClassGPU2(LIFTimeDrivenModel_1_4_GPU2_NEW ** NeuronModel_GPU2, 
		float eexc,float einh,float erest,float vthr,float cm,float tampa,float tnmda,float tinh,float tgj,float tref,
		float grest,float fgj, char const* integrationName, int N_neurons, void ** Buffer_GPU){
	if(blockIdx.x==0 && threadIdx.x==0){
		(*NeuronModel_GPU2) = new LIFTimeDrivenModel_1_4_GPU2_NEW(eexc,einh,erest,vthr,cm,
        tampa,tnmda,tinh,tgj,tref,grest,fgj,integrationName, N_neurons, Buffer_GPU);
	}
}
void LIFTimeDrivenModel_1_4_GPU_NEW::InitializeClassGPU2(int N_neurons){
	hipMalloc(&NeuronModel_GPU2, sizeof(LIFTimeDrivenModel_1_4_GPU2_NEW **));
	
	char * integrationNameGPU;
	hipMalloc((void **)&integrationNameGPU,32*4);
	HANDLE_ERROR(hipMemcpy(integrationNameGPU,integrationMethod_GPU->GetType(),32*4,hipMemcpyHostToDevice));

	this->N_thread = 128;
	this->N_block=prop.multiProcessorCount*16;
	if((N_neurons+N_thread-1)/N_thread < N_block){
		N_block = (N_neurons+N_thread-1)/N_thread;
	}
	int Total_N_thread=N_thread*N_block;

	integrationMethod_GPU->InitializeMemoryGPU(N_neurons, Total_N_thread);

	LIFTimeDrivenModel_1_4_GPU_NEW_InitializeClassGPU2<<<1,1>>>(NeuronModel_GPU2, eexc,einh,erest,vthr,cm,tampa,
		tnmda,tinh,tgj,tref,grest,fgj,integrationNameGPU, N_neurons, integrationMethod_GPU->Buffer_GPU);

	hipFree(integrationNameGPU);
}



__global__ void initializeVectorNeuronState_GPU2(LIFTimeDrivenModel_1_4_GPU2_NEW ** NeuronModel_GPU2, int NumberOfVariables, float * InitialStateGPU, float * AuxStateGPU, float * StateGPU, double * LastUpdateGPU, double * LastSpikeTimeGPU, bool * InternalSpikeGPU, int SizeStates){
	if(blockIdx.x==0 && threadIdx.x==0){
		(*NeuronModel_GPU2)->InitializeVectorNeuronState_GPU2(NumberOfVariables, InitialStateGPU, AuxStateGPU, StateGPU, LastUpdateGPU, LastSpikeTimeGPU, InternalSpikeGPU, SizeStates);
	}
}

void LIFTimeDrivenModel_1_4_GPU_NEW::InitializeVectorNeuronState_GPU2(){
	VectorNeuronState_GPU *state = (VectorNeuronState_GPU *) State;
	initializeVectorNeuronState_GPU2<<<1,1>>>(NeuronModel_GPU2, state->NumberOfVariables, state->InitialStateGPU, state->AuxStateGPU, state->VectorNeuronStates_GPU, state->LastUpdateGPU, state->LastSpikeTimeGPU, state->InternalSpikeGPU, state->SizeStates);
}


__global__ void DeleteClass_GPU2(LIFTimeDrivenModel_1_4_GPU2_NEW ** NeuronModel_GPU2){
	if(blockIdx.x==0 && threadIdx.x==0){
		delete (*NeuronModel_GPU2); 
	}
}


void LIFTimeDrivenModel_1_4_GPU_NEW::DeleteClassGPU2(){
    DeleteClass_GPU2<<<1,1>>>(NeuronModel_GPU2);
    hipFree(NeuronModel_GPU2);
}


bool LIFTimeDrivenModel_1_4_GPU_NEW::CheckSynapseType(Interconnection * connection){
	int Type = connection->GetType();
	if (Type<N_TimeDependentNeuronState && Type >= 0){
		NeuronModel * model = connection->GetSource()->GetNeuronModel();
		//Synapse types that process input spikes 
		if (Type < N_TimeDependentNeuronState && model->GetModelOutputActivityType() == OUTPUT_SPIKE)
			return true;
		else{
			cout << "Synapses type " << Type << " of neuron model " << this->GetTypeID() << ", " << this->GetModelID() << " must receive spikes. The source model generates currents." << endl;
			return false;
		}
		//Synapse types that process input current 
	}
	else{
		cout << "Neuron model " << this->GetTypeID() << ", " << this->GetModelID() << " does not support input synapses of type " << Type << ". Just defined " << N_TimeDependentNeuronState << " synapses types." << endl;
		return false;
	}
}



